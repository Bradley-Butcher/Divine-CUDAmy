
#include <hip/hip_runtime.h>
#include <iostream>

// Kernel definition
__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main() {
    int c; // host copy of c
    int *d_c; // device copy of c
    int size = sizeof(int);

    // Allocate space for device copy of c
    hipMalloc((void **)&d_c, size);

    // Call the kernel
    add<<<1,1>>>(2, 7, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    std::cout << "2 + 7 = " << c << std::endl;

    // Cleanup
    hipFree(d_c);

    return 0;
}